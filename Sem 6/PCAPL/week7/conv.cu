#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define MAX_MASK_WIDTH 5
#define TILE_SIZE 8

__constant__ int M[MAX_MASK_WIDTH];
__host__ void clearMatrix(int *A, int width)
{
	for (int j = 0; j < width; j++)
		A[j] = 0;
}

__global__ void convolution_basic(int* N,int* M, int* P,int mwidth,int width)
{
	int i=blockIdx.x *blockDim.x+threadIdx.x;
	int pv=0;
	int n_start=i-(mwidth/2);
	for(int j=0;j<mwidth;j++)
	{
		if(n_start+j>=0 && n_start+j<width)
			pv+=N[n_start+j]*M[j];
	}
	P[i]=pv;
}

__global__ void convolution_constant(int* N,int* P,int mwidth,int width)
{
	int i=blockIdx.x *blockDim.x+threadIdx.x;
	int pv=0;
	int n_start=i-(mwidth/2);
	for(int j=0;j<mwidth;j++)
	{
		if(n_start+j>=0 && n_start+j<width)
			pv+=N[n_start+j]*M[j];
	}
	P[i]=pv;
}

__global__ void convolution_shared(int *N,int *P,int Mask_Width,int Width)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ int N_ds[TILE_SIZE + MAX_MASK_WIDTH-1];
	int n=(Mask_Width/2);
	int halo_index_left=(blockIdx.x-1)*blockDim.x+threadIdx.x;
	if(threadIdx.x>=blockDim.x-n)
		N_ds[threadIdx.x-(blockDim.x-n)]=(halo_index_left<0)?0:N[halo_index_left];
	N_ds[n+threadIdx.x]=N[blockIdx.x*blockDim.x+threadIdx.x];
	int halo_index_right=(blockIdx.x+1)*blockDim.x+threadIdx.x;
	if(threadIdx.x<n)
		N_ds[n+blockDim.x+threadIdx.x]=(halo_index_right>=Width)?0:N[halo_index_right];
	__syncthreads();
	int Pvalue=0;
	for(int j=0;j<Mask_Width;j++)
		Pvalue+=N_ds[threadIdx.x+j]*M[j];
	P[i]=Pvalue;
}

// __global__ void convolution_shared(int * N, int * P, int mask_width, int width) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     extern __shared__ int N_shared[];
//     // copy to shared memory
//     N_shared[i] = N[i];
//     __syncthreads();
//     int Pvalue = 0;
//     int N_start_point = i - (mask_width / 2);
//     for (int j = 0; j < mask_width; j++) {
//         if (N_start_point + j >= 0 && N_start_point + j < width) {
//             Pvalue += N_shared[N_start_point + j] * M[j];
//         }
//     }
//     P[i] = Pvalue;
// }

int main(void)
{
	hipEvent_t start, stop;
	hipEventCreate( & start);
	hipEventCreate( & stop);
	int n=7;
	int size=n*sizeof(int);
	int* a=(int*)malloc(size);
	int* p=(int*)malloc(size);
	for(int i=0;i<n;i++)
		a[i]=i+1;
	int x=5;
	int size1=x*sizeof(int);
	int* ma=(int*)malloc(size1);
	for(int i=0;i<x;i++)
		ma[i]=rand()%11;
	printf("Original array: \n");
	for(int i=0;i<n;i++)
	printf("%d\n",a[i]);
	printf("Mask array: \n");
	for(int i=0;i<x;i++)
		printf("%d\n",ma[i]);

	int*d_a;
	int* d_ma;
	int* d_p;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_ma, size1);
	hipMalloc((void **)& d_p,size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_ma, ma, size1, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(M),ma,x*sizeof(int));
	hipEventRecord(start);
	convolution_basic<<<1,n>>>(d_a,d_ma,d_p,x,n);
	hipEventRecord(stop);
	hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime( & milliseconds, start, stop);
	printf("Final array from basic:\n");
	for(int i=0;i<n;i++)
		printf("%d\n",p[i]);
	printf("\n");
	printf("Time to taken for 1D convolution kernel for M is %f ms\n",milliseconds);
	clearMatrix(p, size);
	hipEventRecord(start);
	convolution_constant<<<1,x>>>(d_a,d_p,x,n);
	hipEventRecord(stop);
	hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime( & milliseconds, start, stop);
	printf("Final array from constant: \n");
	for(int i=0;i<n;i++)
	printf("%d\n",p[i]);
	printf("\n");
	printf("Time to taken for 1D convolution kernel with constant memory for M is %f ms\n", milliseconds);
	clearMatrix(p, size);

	hipEventRecord(start);
	convolution_shared<<<1,x>>>(d_a,d_p,x,n);
	hipEventRecord(stop);
	hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime( & milliseconds, start, stop);
	printf("Final array from constant: \n");
	for(int i=0;i<n;i++)
	printf("%d\n",p[i]);
	printf("\n");
	printf("Time to taken for 1D convolution kernel with shared memory for M is %f ms\n", milliseconds);
	hipFree(d_a);
	hipFree(d_p);
	hipFree(d_ma);
	return 0;
}
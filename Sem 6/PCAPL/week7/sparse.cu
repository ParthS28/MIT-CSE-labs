#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"


__global__ void func(int num_rows,int *data,int *col_index,int *row_ptr,int *x,int *y)
{
	int row=threadIdx.x;
	if(row<num_rows)
	{
		int dot=0;
		int row_start=row_ptr[row];
		int row_end=row_ptr[row+1];
		for(int i=row_start;i<row_end;i++)
			dot+= data[i]*x[col_index[i]];
		y[row]=dot;
	}
}

int main()
{
	int n;
	printf("Enter size: ");
	scanf("%d",&n);
	int y[n],row_ptr[n+1];
	int ipmat[n][n];
	printf("Enter matrix: \n");
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
			scanf("%d",&ipmat[i][j]);
	}
	int x[n];
	printf("Enter x matrix: \n");
	for(int i=0;i<n;i++)
		scanf("%d",&x[i]);
	int nonzerocount=0;
	for(int i=0;i<n;i++)
	{
		row_ptr[i]=nonzerocount;
		for(int j=0;j<n;j++)
		{
			if(ipmat[i][j]!=0)
				nonzerocount++;
			printf("%d\t",ipmat[i][j]);
		}
		printf("\n");
	}
	row_ptr[n]=nonzerocount;
	int data[nonzerocount],col_index[nonzerocount];
	int k=0;
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			if(ipmat[i][j]!=0)
			{
				data[k]=ipmat[i][j];
				col_index[k++]=j;
			}
		}
	}
	printf("\ndata array\t");
	for(int i=0;i<nonzerocount;i++)
		printf("%d\t",data[i]);
	printf("\ncol_index array\t");
	for(int i=0;i<nonzerocount;i++)
		printf("%d\t",col_index[i]);
	printf("\nrow_ptr array\t");
	for(int i=0;i<=n;i++)
		printf("%d\t",row_ptr[i]);
	printf("\nvector X\t");
	for(int i=0;i<n;i++)
		printf("%d\t",x[i]);
	int *d_data,*d_col_index,*d_row_ptr,*d_x,*d_y;

	hipMalloc((void**)&d_data,nonzerocount*sizeof(int));
	hipMalloc((void**)&d_col_index,nonzerocount*sizeof(int));
	hipMalloc((void**)&d_row_ptr,(n+1)*sizeof(int));
	hipMalloc((void**)&d_x,n*sizeof(int));
	hipMalloc((void**)&d_y,n*sizeof(int));
	hipMemcpy(d_data,data,nonzerocount*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_col_index,col_index,nonzerocount*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr,row_ptr,(n+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);
	
	func<<<1,n>>>(n,d_data,d_col_index,d_row_ptr,d_x,d_y);
	hipMemcpy(y,d_y,n*sizeof(int),hipMemcpyDeviceToHost);
	printf("\nresult\t\t");
	for(int i=0;i<n;i++)
		printf("%d\t",y[i]);
	printf("\n");
	
	hipFree(d_data);
	hipFree(d_col_index);
	hipFree(d_row_ptr);
	hipFree(d_x);
	hipFree(d_y);
	return 0;
}